#include "hip/hip_runtime.h"
#include "Function.cuh"

__global__ void Function()
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > 2)
        return;
    printf("Function %d\n", i);
};
